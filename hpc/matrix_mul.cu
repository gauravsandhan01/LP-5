// matrix_mul.cu

#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void matMul(float *A, float *B, float *C, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < N && col < N) {
        float sum = 0;
        for (int k = 0; k < N; k++)
            sum += A[row * N + k] * B[k * N + col];
        C[row * N + col] = sum;
    }
}

int main() {
    int N = 2;  // 2x2 matrix for simplicity
    size_t size = N * N * sizeof(float);
    float A[] = {1, 2, 3, 4};
    float B[] = {5, 6, 7, 8};
    float C[4];

    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    dim3 threads(16, 16);
    dim3 blocks((N+15)/16, (N+15)/16);
    matMul<<<blocks, threads>>>(d_A, d_B, d_C, N);

    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

    printf("Result Matrix C:\n");
    for (int i = 0; i < N*N; i++) {
        printf("%f ", C[i]);
        if ((i+1) % N == 0) printf("\n");
    }

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    return 0;
}