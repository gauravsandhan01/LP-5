#include <iostream>
#include <hip/hip_runtime.h>

__global__ void add(int *a, int *b, int *c, int n) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < n) c[i] = a[i] + b[i];
}

int main() {
    int n;
    std::cout << "Enter the number of elements: ";
    std::cin >> n;

    size_t size = n * sizeof(int);
    int *a = (int*)malloc(size);
    int *b = (int*)malloc(size);
    int *c = (int*)malloc(size);

    std::cout << "Enter " << n << " elements for array A:\n";
    for (int i = 0; i < n; i++) std::cin >> a[i];

    std::cout << "Enter " << n << " elements for array B:\n";
    for (int i = 0; i < n; i++) std::cin >> b[i];

    int *d_a, *d_b, *d_c;
    hipMalloc(&d_a, size);
    hipMalloc(&d_b, size);
    hipMalloc(&d_c, size);

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    add<<<(n + 255) / 256, 256>>>(d_a, d_b, d_c, n);
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    std::cout << "Result (A + B):\n";
    for (int i = 0; i < n; i++) std::cout << c[i] << " ";
    std::cout << std::endl;

    hipFree(d_a); hipFree(d_b); hipFree(d_c);
    free(a); free(b); free(c);
    return 0;
}
